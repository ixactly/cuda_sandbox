#include <iostream>
#include "Volume.cuh"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "debug.cuh"
#include <unistd.h>

int main() {
    // a

    {// auto vol = new cudaVolume<float> * [3];
        /*
        cuda_ptr<cuda_ptr<cudaVolume<float>>> vol(new cuda_ptr<cudaVolume<float>>[2]);
        // std::cout << sizeof(*vol[0]) << std::endl;

        for (int i = 0; i < 3; i++) {
            // vol[i] = new cudaVolume<float>(10, 10, 10);
            // hipMallocManaged(reinterpret_cast<void **>(&vol[i]), sizeof(cudaVolume<float>));
            vol[i] = cuda_ptr<cudaVolume<float>>(new cudaVolume<float>(100, 100, 100));
            // hipMallocManaged(reinterpret_cast<void**>(&vol[i]), sizeof(cudaVolume<float>));
            vol[i]->init(300, 300, 300);
        }

        vol[1]->printDebug();
        // std::cout << (*vol[1])(1,1,1) << std::endl;
        /*
        vol[1]->forEach([](float val) -> float { return 1.0; });
        std::cout << (*vol[1])(1, 1, 1) << std::endl;

        dim3 block(5, 1, 1);
        dim3 grid(5, 1, 1);

        foo<<<1, 1>>>(vol[1].get());
        hoge<<<1, 1>>>();
        hipDeviceSynchronize();

        std::cout << (*vol[1])(1,1,1) << std::endl;
         */
    }
    {
        /*
        cuda_ptr<float> floating[2];
        for (auto &e : floating) {
            e = cuda_ptr<float>(new float(3.0));
        }
         */
        /*
        cuda_ptr<float> floating = cuda_ptr<float>(new float(4.0f));
        std::cout << *(floating) << std::endl;
         */


        cuda_ptr<float> floating; // nullptr
        auto tmp = new float (5.0f);
        std::cout << "rvalue ptr: " << tmp << std::endl;
        std::cout << "lvalue ptr: " << floating.get() << std::endl;

        floating = cuda_ptr<float>(tmp); // issue

        std::cout << "lvalue after assigned, ptr: " << floating.get() << std::endl;
        // dainyuu to dainyuu constructor de kekka ga tigau
        // cuda_ptr<cudaVolume<float>> single = cuda_ptr<cudaVolume<float>>(new cudaVolume<float>(1000, 1000, 1000));;

        // cuda_ptr<cudaVolume<float>> single;
        // single = cuda_ptr<cudaVolume<float>>(new cudaVolume<float>(1000, 1000, 1000));

        /*
        for (auto &e : single) {
         // e = cuda_ptr<cudaVolume<float>>(new cudaVolume<float>(1000, 1000, 1000));
        }
        */
        hipDeviceSynchronize();
        sleep(2);
    }
    sleep(5);


    return 0;
}
