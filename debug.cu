#include "hip/hip_runtime.h"
//
// Created by tomokimori on 22/09/08.
//
#include <iostream>
#include <unistd.h>
#include "Volume.cuh"

__device__ void bar() {
}

__global__ void foo(CudaVolume<float>* vol) {
    // printf("%d, %d\n", blockIdx.x, threadIdx.x);
    (*vol)(1, 1, 1) = 3.0f;
    // printf("%lf", vol(3, 4, 5));
    bar();
}