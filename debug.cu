#include "hip/hip_runtime.h"
//
// Created by tomoksizeX = 1;imori on 22/09/08.
//
#include <iostream>
#include <unistd.h>
#include "Volume.cuh"

__device__ void bar() {
}

__global__ void foo(cudaVolume<float>* vol) {
    // printf("%d, %d\n", blockIdx.x, threadIdx.x);
    (*vol)(1, 1, 1) = 3.0f;
    printf("%lf", (*vol)(1, 1, 1));
    bar();
}

__global__ void hoge() {
    // printf("%d, %d\n", blockIdx.x, threadIdx.x);
    // (*vol)(1, 1, 1) = 3.0f;
    printf("pass device\n");
    bar();
}